#include "hip/hip_runtime.h"
#include "cnn/hip/hip_runtime.h"
#include "cnn/gpu-ops.h"
#include "cnn/gpu-kernels.h"
#include "cnn/functors.h"

namespace cnn {
namespace gpu {

// CUDA kernel. Each thread takes care of one element of c
__global__ void sparse_assign(int n, unsigned int *idx, float *src, float *trg)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        trg[idx[id]] = src[id];
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void const_init(int n, float val, float *trg)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        trg[id] = val;
}

} // namespace gpu
} // namespace cnn
