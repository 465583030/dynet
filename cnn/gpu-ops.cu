#include "cnn/hip/hip_runtime.h"
#include "cnn/gpu-ops.h"
#include "cnn/gpu-kernels.h"
#include "cnn/functors.h"

namespace cnn {
namespace gpu {

// this wraps kernel dispatches for various operations (preventing us from
// having to compile a version of nodes.cc with NVCC)

void vpairwise_rank_loss(int n, float margin, const float* xgood, const float* xbad, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, xgood, xbad, y, FPairwiseRankLoss(margin));
}

void vpairwise_rank_loss_backward(int n, bool d_wrt_correct, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  if (d_wrt_correct) {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyNegateBackward());
  } else {
    accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
  }
}

void vcwise_product(int n, const float* x0, const float* x1, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  binaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, y, FProduct());
}

void vcwise_product_backward(int n, const float* dEdy, const float* x_other, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, dEdy, x_other, dEdx, FProduct());
}

void vconstant_minusx(int n, float c, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FConstantMinus(c));
}

void vnegate(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FNegate());
}

void vnegate_backward(int n, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accUnaryExprKernel<<<tb.first, tb.second>>>(n, dEdf, dEdx, FNegate());
}

void vrelu(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FRectify());
}

void vrelu_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FRectifyBackward());
}

void vtanh(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FTanh());
}

void vtanh_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FTanhBackward());
}

void vlogistic(int n, const float* x, float* y) {
  auto tb = SizeToBlockThreadPair(n);
  unaryExprKernel<<<tb.first, tb.second>>>(n, x, y, FLogisticSigmoid());
}

void vlogistic_backward(int n, const float* fx, const float* dEdf, float* dEdx) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, fx, dEdf, dEdx, FLogisticSigmoidBackward());
}

void sqeucdist_backward(int n, const float* dEdy, const float* x0, const float* x1, float* dEdx, int i) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, x0, x1, dEdx, FEuclideanBackward(i, dEdy));
}

void sgd_update(int n, const float* g, float* x, float scale, float lambda) {
  auto tb = SizeToBlockThreadPair(n);
  accBinaryExprKernel<<<tb.first, tb.second>>>(n, x, g, x, FL2SGDUpdate(lambda, scale));
}

//// BROKEN STUFF UNDER HERE ///////////////////////////////////

void sqeucdist(int n, const float* x0, const float *x1, float* y) {
  slowReduceKernel<<<1,1>>>(n, x0, x1, y, FSqDist());
}

} // namespace gpu
} // namespace cnn
