#include "hip/hip_runtime.h"
#include "dynet/hip/hip_runtime.h"
#include "dynet/gpu-ops.h"
#include "dynet/gpu-kernels.h"
#include "dynet/functors.h"

namespace dynet {
namespace gpu {

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_const_init(int n, float val, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
    trg[id] = val;
}

void const_init(int n, float val, float *trg) {
  auto tb = SizeToBlockThreadPair(n);
  int total_size = tb.first*tb.second;
  for(int curr_pos = 0; curr_pos < n; curr_pos += total_size)
    ker_const_init<<<tb.first, tb.second>>>(min(total_size, n-curr_pos), val, trg+curr_pos);
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_dense_to_sparse_assign(int n, const unsigned int *idx, float *src, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
    trg[idx[id]] = src[id];
}

void dense_to_sparse_assign(int n, const unsigned int *idx, float *src, float *trg) {
  if(n > 0) {
    auto tb = SizeToBlockThreadPair(n);
    int total_size = tb.first*tb.second;
    for(int curr_pos = 0; curr_pos < n; curr_pos += total_size)
      ker_dense_to_sparse_assign<<<tb.first, tb.second>>>(min(total_size, n-curr_pos), idx+curr_pos, src+curr_pos, trg);
  }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
    trg[id] = src[idx[id]];
}

void sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
  if(n > 0) {
    auto tb = SizeToBlockThreadPair(n);
    int total_size = tb.first*tb.second;
    for(int curr_pos = 0; curr_pos < n; curr_pos += total_size)
      ker_sparse_to_dense_assign<<<tb.first, tb.second>>>(min(total_size, n-curr_pos), idx+curr_pos, src, trg+curr_pos);
  }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n)
    atomicAdd(trg + idx[id], -src[id]);
}

void dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
  if(n > 0) {
    auto tb = SizeToBlockThreadPair(n);
    int total_size = tb.first*tb.second;
    for(int curr_pos = 0; curr_pos < n; curr_pos += total_size)
      ker_dense_to_sparse_subtract<<<tb.first, tb.second>>>(min(total_size, n-curr_pos), idx+curr_pos, src+curr_pos, trg);
  }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n*bsize)
    trg[id] = src[idx[id/bsize]*bsize+id%bsize] * mult;
}

void sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float *src, float *trg) {
  if(n > 0) {
    auto tb = SizeToBlockThreadPair(n*bsize);
    int total_size = tb.first*tb.second;
    for(int curr_pos = 0; curr_pos < n; curr_pos += total_size/bsize)
      ker_sparse_to_dense_block_assign_and_multiply<<<tb.first, tb.second>>>(min(total_size/bsize, n-curr_pos), idx+curr_pos, bsize, mult, src, trg+curr_pos*bsize);
  }
}

// CUDA kernel. Each thread takes care of one element of c
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  // Make sure we do not go out of bounds
  if (id < n*bsize)
    atomicAdd(trg + idx[id/bsize]*bsize+id%bsize, src[id]);
}

void dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float *src, float *trg) {
  if(n > 0) {
    auto tb = SizeToBlockThreadPair(n*bsize);
    int total_size = tb.first*tb.second;
    for(int curr_pos = 0; curr_pos < n; curr_pos += total_size/bsize)
      ker_dense_to_sparse_block_add<<<tb.first, tb.second>>>(min(total_size/bsize, n-curr_pos), idx+curr_pos, bsize, src+curr_pos*bsize, trg);
  }
}

__global__ void PadInputCustomKernelNCHW(float* output, const float* input,
        int N, int C, int H, int W, int pad_right, int pad_bottom) {
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  int idn = id / (W*H*C);
  int idc = (id - idn*W*H*C) / (W*H);
  int idh = (id - idn*W*H*C - idc*W*H) / H;
  int idw = id - idn*W*H*C - idc*W*H - idh*W;
  int out_h = H + pad_bottom;
  int out_w = W + pad_right;
  if (id < N*C*H*W) {
    if (idw < out_w && idh < out_h) {
      output[idn*(C*out_h*out_w) + idc*(out_h*out_w) + idh*(out_w) + idw] = input[id];
    }
  }
}

void pad_input(float* output, const float* input, int N, int C, int H, int W, int pad_right, int pad_bottom) {
  auto tb = SizeToBlockThreadPair(N * C * H * W);
  PadInputCustomKernelNCHW<<<tb.first, tb.second>>>(output, input, N, C, H, W, pad_right, pad_bottom);
}

} // namespace gpu
} // namespace dynet
